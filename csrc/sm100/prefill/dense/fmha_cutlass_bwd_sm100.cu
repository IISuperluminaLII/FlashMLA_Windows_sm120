#include "interface.h"

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_bf16.h>
#include "common/mask.cuh"
#include "common/utils.hpp"
#include "sm100_kernel_traits.hpp"

#include "fmha_cutlass_bwd_sm100.cuh"

template<class Mask, class Varlen, class Element, class ElementOut, class Mla>
void call_run_fmha_bwd([[maybe_unused]] Mask mask, [[maybe_unused]] Varlen is_varlen,
                      [[maybe_unused]] Element in, [[maybe_unused]] ElementOut out, [[maybe_unused]] Mla mla,
                  at::Tensor workspace_buffer, at::Tensor d_o, at::Tensor q, at::Tensor k,
                  at::Tensor v, at::Tensor o, at::Tensor lse,
                  at::Tensor cumulative_seqlen_q, at::Tensor cumulative_seqlen_kv,
                  at::Tensor dq, at::Tensor dk, at::Tensor dv,
                  float softmax_scale, int max_seqlen_q, int total_seqlen_kv) {
  static constexpr bool IsVarlen = std::is_same_v<Varlen, true_type>;
  static constexpr bool IsMla = std::is_same_v<Mla, true_type>;

  // Dual dispatch: Runtime architecture detection for SM100a (server) vs SM120 (workstation)
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  int sm_major = prop.major;
  int sm_minor = prop.minor;
  int sm_version = sm_major * 10 + sm_minor;

  // SM100a = compute_100a (10.0), SM120 = compute_120 (12.0)
  // Runtime guard: Only dispatch if architecture is supported
  if (sm_version >= 120) {
    // Workstation variant: SM120 (RTX 6000 Pro, RTX 50 series)
    using TileShape = std::conditional_t<IsMla,
                                         typename flash::Sm120WorkstationConfig::TileShapeMlaBwd,
                                         typename flash::Sm120WorkstationConfig::TileShapeFmhaBwd>;
    run_fmha_bwd<flash::Sm120WorkstationConfig, Element, IsVarlen, IsMla, TileShape, Mask>(
        workspace_buffer, d_o, q, k, v, o, lse,
        cumulative_seqlen_q, cumulative_seqlen_kv,
        dq, dk, dv,
        softmax_scale, max_seqlen_q, total_seqlen_kv);
  }
#ifndef FLASH_MLA_DISABLE_SM100
  else if (sm_version >= 100) {
    // Server variant: SM100a (B100/B200)
    using TileShape = std::conditional_t<IsMla,
                                         typename flash::Sm100ServerConfig::TileShapeMlaBwd,
                                         typename flash::Sm100ServerConfig::TileShapeFmhaBwd>;
    run_fmha_bwd<flash::Sm100ServerConfig, Element, IsVarlen, IsMla, TileShape, Mask>(
        workspace_buffer, d_o, q, k, v, o, lse,
        cumulative_seqlen_q, cumulative_seqlen_kv,
        dq, dk, dv,
        softmax_scale, max_seqlen_q, total_seqlen_kv);
  }
#endif
  else {
    FLASH_MLA_ASSERT(false && "Unsupported SM architecture: requires SM100a or SM120");
  }
}


void FMHACutlassSM100BwdRun(at::Tensor workspace_buffer, at::Tensor d_o, at::Tensor q, at::Tensor k,
                            at::Tensor v, at::Tensor o, at::Tensor lse,
                            at::Tensor cumulative_seqlen_q, at::Tensor cumulative_seqlen_kv,
                            at::Tensor dq, at::Tensor dk, at::Tensor dv,
                            int mask_mode_code, float softmax_scale, int max_seqlen_q, int max_seqlen_kv, bool is_varlen) {

  const c10::cuda::OptionalCUDAGuard device_guard(q.device());

  int head_dim_qk = q.size(-1);
  int head_dim_vo = v.size(-1);
  MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);
  auto scalar_type_in = q.scalar_type();
  auto scalar_type_out = o.scalar_type();

  if(scalar_type_in == at::ScalarType::BFloat16 && scalar_type_out == at::ScalarType::BFloat16) {
    using Element = cutlass::bfloat16_t;
    using ElementOut = cutlass::bfloat16_t;

    auto apply_config = [&](auto fn) {
      if (mask_mode == MaskMode::kCausal) {
        if(is_varlen) {
          fn(CausalForBackwardMask<false>{}, cute::true_type{}, Element{}, ElementOut{});
        } else {
          fn(CausalForBackwardMask<false>{}, cute::false_type{}, Element{}, ElementOut{});
        }
      }
      else {
        if(is_varlen) {
          fn(ResidualMaskForBackward{}, cute::true_type{}, Element{}, ElementOut{});
        } else {
          fn(ResidualMaskForBackward{}, cute::false_type{}, Element{}, ElementOut{});
        }
      }
    };

    apply_config([&](auto mask, auto varlen, auto in, auto out) {
      if (head_dim_qk == 192 && head_dim_vo == 128) {
        call_run_fmha_bwd(mask, varlen, in, out, true_type{}, workspace_buffer, d_o, q, k, v, o, lse,
                          cumulative_seqlen_q, cumulative_seqlen_kv,
                          dq, dk, dv,
                          softmax_scale, max_seqlen_q, max_seqlen_kv);
      } else if (head_dim_qk == 128 && head_dim_vo == 128) {
        call_run_fmha_bwd(mask, varlen, in, out, false_type{}, workspace_buffer, d_o, q, k, v, o, lse,
                          cumulative_seqlen_q, cumulative_seqlen_kv,
                          dq, dk, dv,
                          softmax_scale, max_seqlen_q, max_seqlen_kv);      }
      else {
        std::cout << "No kernel instantiated for head_dim_qk=" << head_dim_qk << " head_dim_vo=" << head_dim_vo << std::endl;
      }
    });

  } else {
    FLASH_MLA_ASSERT(false);
  }
}
